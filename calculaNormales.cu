#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------*/
/*  FICHERO:       calculaNormales.cu									          */
/*  AUTOR:         Jorge Azorin											  */
/*													                          */
/*  RESUMEN												                      */
/*  ~~~~~~~												                      */
/* Ejercicio grupal para el c�lculo de las normales de una superficie          */
/*----------------------------------------------------------------------------*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>


// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "calculaNormales.h"
#include <Windows.h>



#define ERROR_CHECK { hipError_t err; if ((err = hipGetLastError()) != hipSuccess) { printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

typedef LARGE_INTEGER timeStamp;
double getTime();

/*----------------------------------------------------------------------------*/
/*  FUNCION A PARALELIZAR  (versi�n secuencial-CPU)  				          */
/*	C�lculo de las normales de una superficie definida por una                */
/*  una malla de vtotal x utotal puntos 3D                                    */
/*----------------------------------------------------------------------------*/
int CalculoNormalesCPU()
{
	    TPoint3D direct1, direct2, normal;
		int vecindadU[9]={-1,0,1,1,1,0,-1,-1,-1}; // Vecindad 8 + 1 para calcular todas las rectas
		int vecindadV[9]={-1,-1,-1,0,1,1,1,0,-1};
		int vV,vU;
		int numDir;
		int oKdir1,oKdir2;
		/* La vencidad es:
		*--*--*
		|  |  |
		*--X--*
		|  |  |
		*--*--*
		*/
		int cont=0;

		for (int u = 0; u<S.UPoints; u++)			// Recorrido de todos los puntos de la superficie
		{
			for (int v = 0; v<S.VPoints; v++)
			{
				normal.x=0;
				normal.y=0;
				normal.z=0;
				numDir=0;
				for (int nv = 0; nv < 8 ; nv ++)  // Para los puntos de la vecindad
				{
					    vV=v+vecindadV[nv];
						vU=u+vecindadU[nv];
						if (vV >= 0 && vU >=0 && vV<S.VPoints && vU<S.UPoints)
						{
							direct1.x=S.Buffer[v][u].x-S.Buffer[vV][vU].x;
							direct1.y=S.Buffer[v][u].y-S.Buffer[vV][vU].y;
							direct1.z=S.Buffer[v][u].z-S.Buffer[vV][vU].z;
							oKdir1=1;
						}else
						{
							direct1.x=0.0;
							direct1.y=0.0;
							direct1.z=0.0;
							oKdir1=0;
						}
						vV=v+vecindadV[nv+1];
						vU=v+vecindadU[nv+1];

						if (vV >= 0 && vU >=0 && vV<S.VPoints && vU<S.UPoints)
						{
						   direct2.x=S.Buffer[v][u].x-S.Buffer[vV][vU].x;
						   direct2.y=S.Buffer[v][u].y-S.Buffer[vV][vU].y;
						   direct2.z=S.Buffer[v][u].z-S.Buffer[vV][vU].z;
						   oKdir2=1;
						}else
						{
							direct2.x=0.0;
							direct2.y=0.0;
							direct2.z=0.0;
							oKdir2=0;
						}
						if (oKdir1 ==1 && oKdir2==1)
						{
						  normal.x +=  direct1.y*direct2.z-direct1.z*direct2.y;
						  normal.y += direct1.x*direct2.z-direct1.z*direct2.x;
						  normal.z += direct1.x*direct2.y-direct1.y*direct2.x;
						  numDir++;
						}
				}
				NormalUCPU[cont]=normal.x/(float)numDir;
				NormalVCPU[cont]=normal.y/(float)numDir;
				NormalWCPU[cont]=normal.z/(float)numDir;
				cont++;
			}
		}

	return OKCALC;									// Simulaci�n CORRECTA
}

// ---------------------------------------------------------------
// ---------------------------------------------------------------
// FUNCION A IMPLEMENTAR POR EL GRUPO (paralelizaci�n de CalculoNormalesCPU)
// ---------------------------------------------------------------
// ---------------------------------------------------------------

/*
Esta es la funci�n que tiene que ser implementada por el grupo. 

El objetivo es que sea funcionalmente equivalente a la descrita anteriormente, 
con la diferencia de que los resultados se devolver�n en 
los vectores globales (ya creados en la funci�n principal) NormalUGPU, NormalVGPU y NormalWGPU. 
Como ya se ha dicho, este vector ser� comparado a efectos de correcci�n en la funci�n runTest.

La funci�n debe incluir la gesti�n de memoria para el paso de datos desde la CPU a la GPU y viceversa, 
as� como la llamada a una o varias funciones kernel de CUDA que efect�en la paralelizaci�n 
de las operaciones siguiendo el paradigma SIMD usando la GPU del computador. 
Esta funci�n o funciones kernel deber�n ser definidas por el grupo y se da libertad sobre su contenido y definici�n.
*/

__global__ void calculadorNormales(float* resultado, TSurf aux, char dim) {

	switch (dim) {
	case 'x':

		break;

	case 'y':

		break;

	case 'z':

		break;
	}
}

 int CalculoNormalesGPU()
{
	 TSurf aux;
	 aux.Buffer = S.Buffer;
	 aux.VPoints = S.VPoints;
	 aux.UPoints = S.UPoints;


	 hipMalloc((void**) &aux.UPoints, (sizeof(int) * S.UPoints));
	 hipMalloc((void**)&aux.VPoints, (sizeof(int) * S.VPoints));
	 hipMalloc(&aux.Buffer, (sizeof(TPoint3D) * S.UPoints * S.VPoints));

	 hipMemcpy((void **) aux.UPoints, (void **) S.UPoints, (sizeof(int) * S.UPoints), hipMemcpyHostToDevice);
	 hipMemcpy((void**)aux.VPoints, (void**)S.VPoints, (sizeof(int) * S.VPoints), hipMemcpyHostToDevice);
	 hipMemcpy((void**)aux.Buffer, (void**)S.Buffer, (sizeof(TPoint3D) * S.UPoints * S.VPoints), hipMemcpyHostToDevice);

	 float* d_NormalUGPU = (float*)malloc(S.UPoints * sizeof(int));
	 float* d_NormalVGPU = (float*)malloc(S.VPoints * sizeof(int));
	 float* d_NormalWGPU = (float*)malloc(S.VPoints * S.UPoints * sizeof(int));

	 hipMalloc(&d_NormalUGPU,  S.UPoints * sizeof(int));
	 hipMalloc(&d_NormalVGPU, S.VPoints * sizeof(int));
	 hipMalloc(&d_NormalWGPU, S.VPoints * S.UPoints * sizeof(int));

	 dim3 block(512);
	 dim3 grid( (S.VPoints * S.UPoints +(block.x-1)) / block.x);

	 calculadorNormales<<<grid,block>>>(d_NormalUGPU,aux,'x');
	 hipMemcpy(NormalUGPU, d_NormalUGPU, S.UPoints * sizeof(int), hipMemcpyDeviceToHost);

	 calculadorNormales<<<grid,block>>>(d_NormalVGPU,aux,'y');
	 hipMemcpy(NormalVGPU, d_NormalVGPU, S.VPoints * sizeof(int), hipMemcpyDeviceToHost);

	 calculadorNormales<<<grid,block>>>(d_NormalWGPU,aux,'z');
	 hipMemcpy(NormalWGPU, d_NormalWGPU, S.VPoints * S.UPoints * sizeof(int), hipMemcpyDeviceToHost);

	 return OKCALC;
}
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------

 // Declaraciones adelantadas de funciones
 int LeerSuperficie(const char *fichero);



////////////////////////////////////////////////////////////////////////////////
//PROGRAMA PRINCIPAL
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{


	double gpu_start_time, gpu_end_time;
	double cpu_start_time, cpu_end_time;

	/* Numero de argumentos */
	if (argc != 2)
	{
		fprintf(stderr, "Numero de parametros incorecto\n");
		fprintf(stderr, "Uso: %s superficie\n", argv[0]);
		return;
	}

	/* Apertura de Fichero */
	printf("C�lculo de las normales de la superficie...\n");
	/* Datos de la superficie */
	if (LeerSuperficie((char *)argv[1]) == ERRORCALC)
	{
		fprintf(stderr, "Lectura de superficie incorrecta\n");
		return;
	}
	int numPuntos;
	numPuntos=S.UPoints*S.VPoints;

	// Creaci�n buffer resultados para versiones CPU y GPU
	NormalVCPU = (float*)malloc(numPuntos*sizeof(float));
	NormalUCPU = (float*)malloc(numPuntos*sizeof(float));
    NormalWCPU = (float*)malloc(numPuntos*sizeof(float));
	NormalVGPU = (float*)malloc(numPuntos*sizeof(float));
	NormalUGPU = (float*)malloc(numPuntos*sizeof(float));
	NormalWGPU = (float*)malloc(numPuntos*sizeof(float));

	/* Algoritmo a paralelizar */
	cpu_start_time = getTime();
	if (CalculoNormalesCPU() == ERRORCALC)
	{
		fprintf(stderr, "C�lculo CPU incorrecta\n");
		BorrarSuperficie();
		if (NormalVCPU != NULL) free(NormalVCPU);
		if (NormalUCPU != NULL) free(NormalUCPU);
	    if (NormalWCPU != NULL) free(NormalUCPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		if (NormalWGPU != NULL) free(NormalVGPU);
		if (NormalUGPU != NULL) free(NormalUGPU);		exit(1);
	}
	cpu_end_time = getTime();
	/* Algoritmo a implementar */
	gpu_start_time = getTime();
	if (CalculoNormalesGPU() == ERRORCALC)
	{
		fprintf(stderr, "C�lculo GPU incorrecta\n");
		BorrarSuperficie();
		if (NormalVCPU != NULL) free(NormalVCPU);
		if (NormalUCPU != NULL) free(NormalUCPU);
	    if (NormalWCPU != NULL) free(NormalUCPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		if (NormalUGPU != NULL) free(NormalUGPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		return;
	}
	gpu_end_time = getTime();
	// Comparaci�n de correcci�n
	int comprobar = OKCALC;
	for (int i = 0; i<numPuntos; i++)
	{
		if (((int)NormalVCPU[i]*1000 != (int)NormalVGPU[i])*1000 || ((int)NormalUCPU[i]*1000 != (int)NormalUGPU[i]*1000) || ((int)NormalWCPU[i]*1000 != (int)NormalWGPU[i]*1000))
		{
			comprobar = ERRORCALC;
			fprintf(stderr, "Fallo en el punto %d, valor correcto V=%f U=%f W=%f\n", i, NormalVCPU[i], NormalUCPU[i],NormalWCPU[i]);
		}
	}
	// Impresion de resultados
	if (comprobar == OKCALC)
	{
		printf("C�lculo correcto!\n");

	}
	// Impresi�n de resultados
	printf("Tiempo ejecuci�n GPU : %fs\n", \
		gpu_end_time - gpu_start_time);
	printf("Tiempo de ejecuci�n en la CPU : %fs\n", \
		cpu_end_time - cpu_start_time);
	printf("Se ha conseguido un factor de aceleraci�n %fx utilizando CUDA\n", (cpu_end_time - cpu_start_time) / (gpu_end_time - gpu_start_time));
	// Limpieza de buffers
	BorrarSuperficie();
	if (NormalVCPU != NULL) free(NormalVCPU);
	if (NormalUCPU != NULL) free(NormalUCPU);
    if (NormalWCPU != NULL) free(NormalWCPU);
	if (NormalVGPU != NULL) free(NormalVGPU);
	if (NormalUGPU != NULL) free(NormalUGPU);
	if (NormalWGPU != NULL) free(NormalWGPU);
	return;
}

int
main(int argc, char** argv)
{
	runTest(argc, argv);
	getchar();
}

/* Funciones auxiliares */
double getTime()
{
	timeStamp start;
	timeStamp dwFreq;
	QueryPerformanceFrequency(&dwFreq);
	QueryPerformanceCounter(&start);
	return double(start.QuadPart) / double(dwFreq.QuadPart);
}



/*----------------------------------------------------------------------------*/
/*	Funci�n:  LeerSuperficie(char *fichero)						              */
/*													                          */
/*	          Lee los datos de la superficie de un fichero con formato .FOR   */
/*----------------------------------------------------------------------------*/
int LeerSuperficie(const char *fichero)
{
	int i, j, count;		/* Variables de bucle */
	int utotal,vtotal;		/* Variables de tama�o de superficie */
	FILE *fpin; 			/* Fichero */
	double x, y, z;

	/* Apertura de Fichero */
	if ((fpin = fopen(fichero, "r")) == NULL) return ERRORCALC;
	/* Lectura de cabecera */
	if (fscanf(fpin, "Ancho=%d\n", &utotal)<0) return ERRORCALC;
	if (fscanf(fpin, "Alto=%d\n", &vtotal)<0) return ERRORCALC;
	if (utotal*vtotal <= 0) return ERRORCALC;
	/* Localizacion de comienzo */
	if (feof(fpin)) return ERRORCALC;
	/* Inicializaci�n de parametros geometricos */
	if (CrearSuperficie(utotal, vtotal) == ERRORCALC) return ERRORCALC;
	/* Lectura de coordenadas */
	count = 0;
	for (i = 0; i<utotal; i++)
	{
		for (j = 0; j<vtotal; j++)
		{
			if (!feof(fpin))
			{
				fscanf(fpin, "%lf %lf %lf\n", &x, &y, &z);
				S.Buffer[j][i].x = x;
				S.Buffer[j][i].y = y;
				S.Buffer[j][i].z = z;
				count++;
			}
			else break;
		}
	}
	fclose(fpin);
	if (count != utotal*vtotal) return ERRORCALC;
	return OKCALC;
}



